#include "hip/hip_runtime.h"
/* Copyright 2015 The math21 Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include "inner.h"
#include "../operations.h"

namespace math21 {

#define GPU_BLOCK_SIZE 16

    // matrix multiplication
    template<typename T>
    __global__ void gpu_matrix_multiply_easy(NumR s, T *a, T *b, T *c, int n, int r, int m) {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        T sum = 0;
        if (row < n && col < m) {
            for (int i = 0; i < r; i++) {
                sum += a[row * r + i] * b[i * m + col];
            }
            c[row * m + col] = (T) s * sum;
        }
    }

    // block matrix multiplication
    template<typename T>
    __global__ void gpu_matrix_multiply_shared(NumR s, T *A, T *B, T *C,
                                               int nr_A, int nc_A, int nc_B) {
        __shared__ T sA[GPU_BLOCK_SIZE][GPU_BLOCK_SIZE];   // Tile size to store elements in shared memory
        __shared__ T sB[GPU_BLOCK_SIZE][GPU_BLOCK_SIZE];

        int row = blockDim.y * blockIdx.y + threadIdx.y; //To generate ids of threads.
        int col = blockDim.x * blockIdx.x + threadIdx.x;
        T tmp = 0;

        for (int k = 0; k < (((nc_A - 1) / GPU_BLOCK_SIZE) + 1); k++) {
            if ((row < nr_A) && (threadIdx.x + (k * GPU_BLOCK_SIZE)) < nc_A) {
                sA[threadIdx.y][threadIdx.x] = A[(row * nc_A) + threadIdx.x + (k * GPU_BLOCK_SIZE)];
            } else {
                sA[threadIdx.y][threadIdx.x] = 0;
            }
            if (col < nc_B && (threadIdx.y + k * GPU_BLOCK_SIZE) < nc_A) {
                sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k * GPU_BLOCK_SIZE) * nc_B + col];
            } else {
                sB[threadIdx.y][threadIdx.x] = 0;
            }
            __syncthreads();

            for (int j = 0; j < GPU_BLOCK_SIZE; ++j) {
                tmp += sA[threadIdx.y][j] * sB[j][threadIdx.x];
            }
            __syncthreads();
        }
        if (row < nr_A && col < nc_B) {
            C[row * nc_B + col] = (T) s * tmp;
        }
    }

    // MATH21_ASSERT(A.isContinuous() && !A.isColumnMajor());
    // MATH21_ASSERT(B.isContinuous() && !B.isColumnMajor());
    template<typename T>
    void _math21_c_matrix_multiply_cuda(NumR s, const Tensor<T> &A, const Tensor<T> &B, Tensor<T> &C) {
        MATH21_ASSERT(!A.isEmpty() && !B.isEmpty(), "empty matrix");
        MATH21_ASSERT(B.nrows() == A.ncols(), "matrix size doesn't match in *");

        NumN n, m, r;
        n = A.nrows();
        m = B.ncols();
        r = A.ncols();
        if (C.nrows() != n || C.ncols() != m) {
            if (m == 1) {
                C.setSize(n);
            } else {
                C.setSize(n, m);
            }
        }
        MATH21_ASSERT(C.isContinuous() && !C.isColumnMajor());

        const T *A_data = math21_memory_tensor_data_address(A);
        const T *B_data = math21_memory_tensor_data_address(B);
        T *C_data = math21_memory_tensor_data_address(C);

        T *d_a, *d_b, *d_c;
        math21_cuda_malloc_device((void **) &d_a, sizeof(T) * n * r);
        math21_cuda_malloc_device((void **) &d_b, sizeof(T) * r * m);
        math21_cuda_malloc_device((void **) &d_c, sizeof(T) * n * m);

        // copy matrix A and B from host to device memory
        math21_cuda_memcpy_host_to_device(d_a, A_data, sizeof(T) * n * r);
        math21_cuda_memcpy_host_to_device(d_b, B_data, sizeof(T) * r * m);

        unsigned int grid_rows = (unsigned int) (n + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE;
        unsigned int grid_cols = (unsigned int) (m + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE;
        dim3 dimGrid(grid_cols, grid_rows, 1);
        dim3 dimBlock(GPU_BLOCK_SIZE, GPU_BLOCK_SIZE, 1);

//        timer t;
//        t.start();

        // Launch kernel
#ifdef MATH21_FLAG_UNDERSTANDABLE
        gpu_matrix_multiply_easy << < dimGrid, dimBlock >> > (s, d_a, d_b, d_c, n, r, m);
#else
        gpu_matrix_multiply_shared << < dimGrid, dimBlock >> > (s, d_a, d_b, d_c, n, r, m);
#endif

        // Transefr results from device to host
        math21_cuda_memcpy_device_to_host(C_data, d_c, sizeof(T) * n * m);
        math21_cuda_DeviceSynchronize();

        math21_cuda_free_device(d_a);
        math21_cuda_free_device(d_b);
        math21_cuda_free_device(d_c);

//        t.end();
//        printf("Time elapsed %f ms.\n\n", t.time());
    }

    namespace detail {
        void _math21_c_matrix_multiply_cuda_Num(NumR s, const TenN &A, const TenN &B, TenN &C) {
            _math21_c_matrix_multiply_cuda(s, A, B, C);
        }

        void _math21_c_matrix_multiply_cuda_Num(NumR s, const TenZ &A, const TenZ &B, TenZ &C) {
            _math21_c_matrix_multiply_cuda(s, A, B, C);
        }

        void _math21_c_matrix_multiply_cuda_Num(NumR s, const TenR &A, const TenR &B, TenR &C) {
            _math21_c_matrix_multiply_cuda(s, A, B, C);
        }

        void _math21_c_matrix_multiply_cuda_Num(NumR s, const Tensor<int> &A, const Tensor<int> &B, Tensor<int> &C) {
            _math21_c_matrix_multiply_cuda(s, A, B, C);
        }
    }
}